#include "hip/hip_runtime.h"
#include <cub/warp/warp_reduce.cuh>
#include <hipcub/hipcub.hpp>

#define num_values_4bit 32
template <typename T, int THREADS, int BITS> __global__ void kgemm_4bit_inference_naive(int M, int N, int K, T * __restrict__ const A, unsigned char *B,  float *absmax, const float *datatype, T * out,  int lda, int ldb, int ldc, int blocksize)
{
  // per threadblock:
  // load step-by-step in chunks of [32,warps]: 1x32 * [32,warps] -> [1,warps]
  // 4 warps -> 4 loads per iter
  // 1x32 * 32x4 -> 1x4 outputs per thread block
  typedef hipcub::WarpReduce<float> WarpReduce;
  __shared__ typename WarpReduce::TempStorage temp_storage[THREADS/32];

  const int warp_idx = threadIdx.x / 32;
  const int warp_lane = threadIdx.x % 32;
  const int row_B = (THREADS/32)*blockIdx.x + warp_idx;
  const int num_values_8bit = num_values_4bit/2;
  float local_C = 0.0f;

  unsigned char local_B_4bit[num_values_8bit];
  T local_B[num_values_4bit/4];
  T local_A[num_values_4bit/4];
  __shared__ T quant_map[16];
	T local_absmax = T(0.0f);

  for(int i = threadIdx.x; i < 16; i++)
    quant_map[i] = T(datatype[i]);
  __syncthreads();

  // A: [1, K]
  // B: [N, K]
  for(int inner_idx = warp_lane*num_values_4bit; inner_idx < K; inner_idx += 32*num_values_4bit)
  {
    int inner_idx_halved = inner_idx/2;
    int offset_B = ldb*row_B;
    int absidx = ((2*offset_B)+inner_idx)/blocksize;
	  local_absmax = __ldg(&(absmax[absidx]));

    if(row_B < M)
    {
      if((inner_idx_halved + num_values_8bit) < (K/2))
      {
        // this is the most important for performance considerations
        reinterpret_cast<int4(&)[num_values_8bit]>(local_B_4bit)[0] = reinterpret_cast<int4*>(B)[(offset_B+(inner_idx_halved))/(num_values_8bit)];
      }
      else
      {
        #pragma unroll
        for(int j = 0; j < (num_values_8bit); j++)
          if((inner_idx_halved) + j < (K/2))
            local_B_4bit[j] = B[offset_B+inner_idx_halved + j];
          else
            local_B_4bit[j] = 0b01110111;
      }
    }
    else
    {
      #pragma unroll
      for(int j = 0; j < (num_values_8bit); j++)
          local_B_4bit[j] = 0b01110111;
    }

    for(int i = 0; i < 4; i++)
    {
      #pragma unroll
      for(int k = 0; k < num_values_8bit/4; k++)
      {
        #if __CUDA_ARCH__ >= 800
          local_B[k*2] = quant_map[local_B_4bit[(i*num_values_8bit/4) + k] >> 4]*local_absmax;
          local_B[k*2 + 1] = quant_map[local_B_4bit[(i*num_values_8bit/4) + k] & 0x0F]*local_absmax;
        #else
          // bf16 multipliation not supported
          local_B[k*2] = T((float)quant_map[local_B_4bit[(i*num_values_8bit/4) + k] >> 4]*(float)local_absmax);
          local_B[k*2 + 1] = T((float)quant_map[local_B_4bit[(i*num_values_8bit/4) + k] & 0x0F]*(float)local_absmax);
        #endif
      }

      if(inner_idx+(num_values_4bit/4) + (i*num_values_4bit/4) < K)
      {
        // this is also relatively important for performance
        if(BITS==16)
        {
          reinterpret_cast<int4(&)[num_values_4bit]>(local_A)[0] = reinterpret_cast<int4*>(A)[inner_idx/(num_values_4bit/4) + i];
        }
        else
        {
          reinterpret_cast<int4(&)[num_values_4bit]>(local_A)[0] = reinterpret_cast<int4*>(A)[inner_idx/(num_values_4bit/8) + (2*i) + 0];
          reinterpret_cast<int4(&)[num_values_4bit]>(local_A)[1] = reinterpret_cast<int4*>(A)[inner_idx/(num_values_4bit/8) + (2*i) + 1];
        }

      }
      else
        #pragma unroll
        for(int k = 0; k < num_values_4bit/4; k++)
          if(inner_idx + (i*num_values_4bit/4) + k < K)
            local_A[k] = A[inner_idx + k + (i*num_values_4bit/4)];
          else
            local_A[k] = T(0.0f);


      // accumulate in float; small performance hit for Ampere, but lower error for outputs
      #pragma unroll
      for(int k = 0; k < num_values_4bit/4; k++)
      {
        #if __CUDA_ARCH__ >= 800
          local_C += (float)(local_A[k]*local_B[k]);
        #else
          // bf16 multipliation not supported
          local_C += ((float)local_A[k]*(float)local_B[k]);
        #endif
      }
    }
  }

  local_C = WarpReduce(temp_storage[warp_idx]).Sum(local_C);

  if(row_B < M && warp_lane == 0)
    out[row_B] = T(local_C);

}

template <typename T, int BITS> void gemm_4bit_inference_naive(int m, int n, int k, T * A,  unsigned char* B,  float *absmax, float *datatype, T * out,  int lda, int ldb, int ldc, int blocksize, hipStream_t stream)
{

  int num_blocks = (m+3)/4;
  kgemm_4bit_inference_naive<T, 128, BITS><<< num_blocks, 128, 0, stream>>>(m,  n,  k, A,  B, absmax, datatype, out, lda, ldb, ldc, blocksize);
}

extern "C" void gemm_4bit_inference_naive_f16(int m, int n, int k, half * A,  unsigned char* B,  float *absmax, float *code, half * out,  int lda, int ldb, int ldc, int blocksize, hipStream_t stream)
{ gemm_4bit_inference_naive<half, 16>(m, n, k, A, B, absmax,  code, out, lda, ldb, ldc, blocksize, stream); }

extern "C" void gemm_4bit_inference_naive_bf16(int m, int n, int k, __hip_bfloat16 * A,  unsigned char* B,  float *absmax, float *code, __hip_bfloat16 * out,  int lda, int ldb, int ldc, int blocksize, hipStream_t stream)
{ gemm_4bit_inference_naive<__hip_bfloat16, 16>(m, n, k, A, B, absmax,  code, out, lda, ldb, ldc, blocksize, stream); }

extern "C" void gemm_4bit_inference_naive_f32(int m, int n, int k, float * A,  unsigned char* B,  float *absmax, float *code, float * out,  int lda, int ldb, int ldc, int blocksize, hipStream_t stream)
{ gemm_4bit_inference_naive<float, 32>(m, n, k, A, B, absmax,  code, out, lda, ldb, ldc, blocksize, stream); }

